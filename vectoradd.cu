#include "hip/hip_runtime.h"
/* 
   * This example explains how to divide the host and 
   * device code into separate files using vector addition 
*/
#include "kernel.h"
#define N 64



__global__ void addKernel(float *a,float *b) {
	int idx=threadIdx.x+blockIdx.x*blockDim.x;

	if(idx>=N) return;
		a[idx]+=b[idx];
}


void vectorAdd() {
	//host memory
	float *h_arr1,*h_arr2,*h_res;
	size_t size=N*sizeof(float);

	//allocate host memory
	h_arr1=(float*)malloc(size);
	h_arr2=(float*)malloc(size);
	h_res=(float*)malloc(size);

	//populate the host arrays
	for(int i=0;i<N;i++) {
		h_arr1[i]=i+1;
		h_arr2[i]=i+2;
	}

	//device memory
	float *d_arr1,*d_arr2;

	//allocate device memory
	hipMalloc((void**)&d_arr1,size);
	hipMalloc((void**)&d_arr2,size);

	//copy host to device
	hipMemcpy(d_arr1,h_arr1,size,hipMemcpyHostToDevice);
	hipMemcpy(d_arr2,h_arr2,size,hipMemcpyHostToDevice);

	addKernel<<<8,8>>>(d_arr1,d_arr2);


	//copy result device to host
	hipMemcpy(h_res,d_arr1,size,hipMemcpyDeviceToHost);

	//print result array
	for(int i=10;i<20;i++)
		cout << h_arr1[i] << " ";
	cout << endl;

	for(int i=10;i<20;i++)
		cout << h_arr2[i] << " ";
	cout << endl;

	for(int i=10;i<20;i++)
		cout << h_res[i] << " ";
	cout << endl;

	//free host and device memory
	free(h_arr1); free(h_arr2); free(h_res);
	hipFree(d_arr1); hipFree(d_arr2);
}
